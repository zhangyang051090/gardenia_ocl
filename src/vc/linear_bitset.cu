#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu> and Pingfan Li <lipingfan@163.com>
#define VC_VARIANT "linear_bitset"
#include <hipcub/hipcub.hpp>
#include "vc.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "worklistc.h"

__device__ __forceinline__ void assignColor(unsigned int *forbiddenColors, int *colors, int node) {
	int i;
/*
	int c = 32;
	for (i = 0; i < MAXCOLOR/32; i++) {
		if (forbiddenColors[i] != 0) {
			forbiddenColors[i] &= -(signed)forbiddenColors[i];
			if (forbiddenColors[i]) c--;
			if (forbiddenColors[i] & 0x0000ffff) c -= 16;
	        	if (forbiddenColors[i] & 0x00ff00ff) c -= 8;
        		if (forbiddenColors[i] & 0x0f0f0f0f) c -= 4;
		        if (forbiddenColors[i] & 0x33333333) c -= 2;
        		if (forbiddenColors[i] & 0x55555555) c -= 1;
			colors[node] = c;
			break;
		}
		else
			c += 32;
	}
//*/
///*
	for (i = 0; i < MAXCOLOR/32; i++) {
		int pos = __ffs(forbiddenColors[i]);
		if(pos) {
			colors[node] = i * 32 + pos - 1;
			break;
		}
	}
	assert(i < MAXCOLOR/32);
//*/
}

__global__ void first_fit(int m, int *row_offsets, int *column_indices, Worklist2 inwl, int *colors) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned forbiddenColors[MAXCOLOR/32+1];
	int vertex;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int j = 0; j < MAXCOLOR/32; j++)
			forbiddenColors[j] = 0xffffffff;
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			int color = colors[neighbor];
			forbiddenColors[color / 32] &= ~(1 << (color % 32));
		}
		assignColor(forbiddenColors, colors, vertex);
	}
}

__global__ void conflict_resolve(int m, int *row_offsets, int *column_indices, Worklist2 inwl, Worklist2 outwl, int *colors) {
	//typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex;
	int conflicted = 0;
	if (inwl.pop_id(id, vertex)) {
		int row_begin = row_offsets[vertex];
		int row_end = row_offsets[vertex + 1];
		for (int offset = row_begin; offset < row_end; offset ++) {
			int neighbor = column_indices[offset];
			if (colors[vertex] == colors[neighbor] && vertex < neighbor) {
				conflicted = 1;
				colors[vertex] = MAXCOLOR;
				break;
			}
		}
	}
	//outwl.push_1item<BlockScan>(conflicted, vertex, BLOCK_SIZE);
	if(conflicted) outwl.push(vertex);
}

int VCSolver(int m, int nnz, int *row_offsets, int *column_indices, int *colors) {
	int num_colors = 0, iter = 0;
	int *d_row_offsets, *d_column_indices, *d_colors;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_colors, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_colors, colors, m * sizeof(int), hipMemcpyHostToDevice));

	int nitems = m;
	Worklist2 inwl(m), outwl(m);
	Worklist2 *inwlptr = &inwl, *outwlptr = &outwl;
	for(int i = 0; i < m; i ++) inwl.h_queue[i] = i;
	inwl.set_index(m);
	CUDA_SAFE_CALL(hipMemcpy(inwl.d_queue, inwl.h_queue, m * sizeof(int), hipMemcpyHostToDevice));
	//thrust::sequence(thrust::device, inwl.d_queue, inwl.d_queue + m);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	printf("Launching CUDA VC solver (%d threads/CTA) ...\n", BLOCK_SIZE);

	Timer t;
	t.Start();
	while (nitems > 0) {
		iter ++;
		int nblocks = (nitems - 1) / BLOCK_SIZE + 1;
		first_fit<<<nblocks, BLOCK_SIZE>>>(m, d_row_offsets, d_column_indices, *inwlptr, d_colors);
		conflict_resolve<<<nblocks, BLOCK_SIZE>>>(m, d_row_offsets, d_column_indices, *inwlptr, *outwlptr, d_colors);
		nitems = outwlptr->nitems();
		Worklist2 * tmp = inwlptr;
		inwlptr = outwlptr;
		outwlptr = tmp;
		outwlptr->reset();
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	CUDA_SAFE_CALL(hipMemcpy(colors, d_colors, m * sizeof(int), hipMemcpyDeviceToHost));
	#pragma omp parallel for reduction(max : num_colors)
	for (int n = 0; n < m; n ++)
		num_colors = max(num_colors, colors[n]);
	num_colors ++;
    printf("\titerations = %d.\n", iter);
    printf("\truntime[%s] = %f ms, num_colors = %d.\n", VC_VARIANT, t.Millisecs(), num_colors);
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_colors));
	return num_colors;
}

