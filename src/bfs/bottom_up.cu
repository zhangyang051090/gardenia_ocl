#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Author: Xuhao Chen <cxh@illinois.edu>
#define BFS_VARIANT "bottom_up"
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include "bfs.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

__global__ void bottom_up_kernel(int m, int *row_offsets, int *column_indices, DistT *depths, bool *changed, bool *front, bool *next, int depth) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src = tid;
	if(src < m && depths[src] == MYINFINITY) { // not visited
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if(front[dst]) { // if the parent is in the current frontier
				//atomicAdd(frontier_size, 1);
				depths[src] = depths[dst] + 1;
				//depths[src] = depth;
				next[src] = true; // put this vertex into the next frontier
				*changed = true;
			}
		}
	}
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *out_row_offsets, int *out_column_indices, int *h_degree, DistT *h_depths) {
	//print_device_info(0);
	DistT zero = 0;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, in_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, in_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_depths;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_depths, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_depths, h_depths, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_depths[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	bool *d_changed, h_changed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	//int *d_num_frontier;
	//CUDA_SAFE_CALL(hipMalloc((void **)&d_num_frontier, sizeof(int)));
	bool *front, *next;
	CUDA_SAFE_CALL(hipMalloc((void **)&front, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&next, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(front, 0, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(next, 0, m * sizeof(bool)));
	thrust::fill(thrust::device, front + source, front + source + 1, 1); // set the source vertex

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	//int h_num_frontier = 1;

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		//CUDA_SAFE_CALL(hipMemcpy(d_num_frontier, &zero, sizeof(int), hipMemcpyHostToDevice));
		bottom_up_kernel <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_depths, d_changed, front, next, iter);
		CudaTest("solving failed");
		// swap the queues
		bool *temp = front;
		front = next;
		next = temp;
		thrust::fill(thrust::device, next, next + m, 0);
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(&h_num_frontier, d_num_frontier, sizeof(int), hipMemcpyDeviceToHost));
		//printf("iteration=%d, num_frontier=%d\n", iter, h_num_frontier);
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());

	CUDA_SAFE_CALL(hipMemcpy(h_depths, d_depths, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_depths));
	CUDA_SAFE_CALL(hipFree(d_changed));
	//CUDA_SAFE_CALL(hipFree(d_num_frontier));
	return;
}
