#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Author: Xuhao Chen <cxh@illinois.edu>
#define BFS_VARIANT "linear_base"
#include "bfs.h"
#include "worklistc.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

__global__ void bfs_kernel(int m, int *row_offsets, int *column_indices, DistT *dist, Worklist2 in_queue, Worklist2 out_queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if(in_queue.pop_id(tid, src)) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];

			//DistT new_dist = dist[src] + 1;
			if ((dist[dst] == MYINFINITY) && (atomicCAS(&dist[dst], MYINFINITY, dist[src]+1)==MYINFINITY)) {
			//if (dist[dst] == MYINFINITY) {//Not visited
			//	dist[dst] = new_dist;
				assert(out_queue.push(dst));
			}
		}
	}
}

__global__ void insert(int source, Worklist2 queue) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == 0) queue.push(source);
	return;
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degree, DistT *h_dist) {
	//print_device_info(0);
	DistT zero = 0;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(zero), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	Worklist2 queue1(m), queue2(m);
	Worklist2 *in_frontier = &queue1, *out_frontier = &queue2;
	int iter = 0;
	int nitems = 1;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA BFS solver (%d threads/CTA) ...\n", nthreads);
	printf("row_offsets[0] %d) ...\n", h_row_offsets[0]);
	printf("row_offsets[1] %d) ...\n", h_row_offsets[1]);

	Timer t;
	t.Start();
	insert<<<1, nthreads>>>(source, *in_frontier);
	nitems = in_frontier->nitems();
	printf("nitems_1 is %d) ...\n", nitems);
		
	do {
		++ iter;
		nblocks = (nitems - 1) / nthreads + 1;
		//printf("iteration %d: frontier_size = %d\n", iter, nitems);
		bfs_kernel <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_dist, *in_frontier, *out_frontier);
		CudaTest("solving failed");
		nitems = out_frontier->nitems();
		Worklist2 *tmp = in_frontier;
		in_frontier = out_frontier;
		out_frontier = tmp;
		out_frontier->reset();
		printf("nitems is %d) ...\n", nitems);
	} while (nitems > 0);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	return;
}
