#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define BFS_VARIANT "topo_lb"
#include "bfs.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
#include "timer.h"

typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__device__ __forceinline__ void process_edge(int depth, int edge, int *column_indices, DistT *dist, bool *changed) {
	int dst = column_indices[edge];
	//assert(dst < m);
	if (dist[dst] > depth) {
	//if(dist[dst] == MYINFINITY) {
		dist[dst] = depth;
		*changed = true;
	}
}

__device__ void expandByCta(int m, int *row_offsets, int *column_indices, DistT *dist, bool *visited, bool *expanded, int depth, bool *changed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex = id;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	if(vertex < m && visited[vertex] && !expanded[vertex]) {
		size = row_offsets[vertex + 1] - row_offsets[vertex];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1)
			break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = vertex;
			expanded[id] = true;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_vertex];
		int row_end = row_offsets[sh_vertex + 1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				process_edge(depth, edge, column_indices, dist, changed);
			}
		}
	}
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByWarp(int m, int *row_offsets, int *column_indices, DistT *dist, bool *visited, bool *expanded, int depth, bool *changed) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	int vertex = id;
	if(vertex < m && visited[vertex] && !expanded[vertex]) {
		size = row_offsets[vertex + 1] - row_offsets[vertex];
	}
	while(__any(size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = vertex;
			expanded[id] = true;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner + 1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				process_edge(depth, edge, column_indices, dist, changed);
			}
		}
	}
}

__global__ void bfs_kernel(int m, int *row_offsets, int *column_indices, DistT *dist, bool *changed, bool *visited, bool *expanded, int *frontier_size, int depth) {
	expandByCta(m, row_offsets, column_indices, dist, visited, expanded, depth, changed);
	expandByWarp(m, row_offsets, column_indices, dist, visited, expanded, depth, changed);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src = tid;
	const int SCRATCHSIZE = BLOCK_SIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	gather_offsets[threadIdx.x] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(src < m && visited[src] && !expanded[src]) { // visited but not expanded
		expanded[src] = true;
		neighbor_offset = row_offsets[src];
		neighbor_size = row_offsets[src+1] - neighbor_offset;
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		int edge = gather_offsets[threadIdx.x];
		if(threadIdx.x < total_edges) {
			process_edge(depth, edge, column_indices, dist, changed);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

__global__ void bfs_update(int m, DistT *dist, bool *visited) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(dist[id] < MYINFINITY && !visited[id])
			visited[id] = true;
	}
}

void BFSSolver(int m, int nnz, int source, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degree, DistT *h_dist) {
	//print_device_info(0);
	DistT zero = 0;
	bool one = 1;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(DistT), hipMemcpyHostToDevice));
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(&d_visited[source], &one, sizeof(bool), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(d_expanded, 0, m * sizeof(bool)));
	int *d_frontier_size;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_frontier_size, sizeof(int)));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	//int h_frontier_size = 1;
	printf("Launching CUDA BFS solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		//CUDA_SAFE_CALL(hipMemcpy(d_frontier_size, &zero, sizeof(int), hipMemcpyHostToDevice));
		bfs_kernel <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_dist, d_changed, d_visited, d_expanded, d_frontier_size, iter);
		bfs_update <<<nblocks, nthreads>>> (m, d_dist, d_visited);
		CudaTest("solving failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(&h_frontier_size, d_frontier_size, sizeof(int), hipMemcpyDeviceToHost));
		//printf("iteration=%d, frontier_size=%d\n", iter, h_frontier_size);
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", BFS_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_changed));
	CUDA_SAFE_CALL(hipFree(d_visited));
	CUDA_SAFE_CALL(hipFree(d_expanded));
	CUDA_SAFE_CALL(hipFree(d_frontier_size));
	return;
}
