#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define PR_VARIANT "gather_warp"
#include <hipcub/hipcub.hpp>
#include "pr.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

__global__ void calc_contrib(int m, ScoreT *scores, int *degree, ScoreT *outgoing_contrib) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if (src < m) outgoing_contrib[src] = scores[src] / degree[src];
}

// gather operation needs incoming neighbor list
__global__ void gather(int m, int *row_offsets, int *column_indices, ScoreT *scores, ScoreT *contrib, float *diff, const ScoreT base_score) {
	typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	__shared__ ScoreT sdata[BLOCK_SIZE + 16];                       // padded to avoid reduction ifs
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];

	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	float error = 0;
	for(int src = warp_id; src < m; src += num_warps) {
		// use two threads to fetch row_offsets[src] and row_offsets[src+1]
		// this is considerably faster than the straightforward version
		if(thread_lane < 2)
			ptrs[warp_lane][thread_lane] = row_offsets[src + thread_lane];
		const int row_begin = ptrs[warp_lane][0];                   //same as: row_begin = row_offsets[isrc];
		const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = row_offsets[src+1];

		// compute local sum
		ScoreT sum = 0;
		for (int offset = row_begin + thread_lane; offset < row_end; offset += WARP_SIZE) {
			int dst = column_indices[offset];
			sum += contrib[dst];
		}
		// store local sum in shared memory
		sdata[threadIdx.x] = sum; __syncthreads();

		// reduce local sums to row sum
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();

		if(thread_lane == 0) {
			ScoreT old_score = scores[src];
			ScoreT new_score = base_score + kDamp * sdata[threadIdx.x];
			scores[src] = new_score;
			error += fabs(new_score - old_score);
		}
	}
	float block_sum = BlockReduce(temp_storage).Sum(error);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

void PRSolver(int m, int nnz, int *h_row_offsets, int *h_column_indices, int *out_row_offsets, int *out_column_indices, int *h_degree, ScoreT *h_scores) {
	//print_device_info(0);
	int *d_row_offsets, *d_column_indices, *d_degree;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_degree, m * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_degree, h_degree, m * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, h_scores, m * sizeof(ScoreT), hipMemcpyHostToDevice));
	ScoreT *d_contrib;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_contrib, m * sizeof(ScoreT)));
	float *d_errors;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_errors, m * sizeof(float)));
	float *d_diff, h_diff;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));

	int iter = 0;
	const ScoreT base_score = (1.0f - kDamp) / m;
	const int nthreads = BLOCK_SIZE;
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	const int nSM = deviceProp.multiProcessorCount;
	const int max_blocks_per_SM = maximum_residency(gather, nthreads, 0);
	const int max_blocks = max_blocks_per_SM * nSM;
	const int nblocks = std::min(max_blocks, DIVIDE_INTO(m, WARPS_PER_BLOCK));
	printf("Launching CUDA PR solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		calc_contrib<<<(m - 1) / nthreads + 1, nthreads>>>(m, d_scores, d_degree, d_contrib);
		CudaTest("solving kernel calc_contrib failed");
		//CUDA_SAFE_CALL(hipMemset(d_errors, 0, m * sizeof(float)));
		gather<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_scores, d_contrib, d_diff, base_score);
		CudaTest("solving kernel gather failed");
		//h_diff = thrust::reduce(thrust::device, d_errors, d_errors+m);
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		//printf("iteration=%d, diff=%f\n", iter, h_diff);
		printf(" %2d    %f\n", iter, h_diff);
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_degree));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_contrib));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
