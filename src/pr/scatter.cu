#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define PR_VARIANT "scatter"
#include "pr.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;

__global__ void initialize(int m, ScoreT *next_scores, ScoreT base_score) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) next_scores[id] = base_score;
}

__global__ void scatter(int m, int *row_offsets, int *column_indices, ScoreT *cur_scores, ScoreT *next_scores) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
		int degree = row_end - row_begin;
		ScoreT value = kDamp * cur_scores[src] / (ScoreT)degree;
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			atomicAdd(&next_scores[dst], value);
		}
	}
}

__global__ void reduce(int m, ScoreT *cur_scores, ScoreT *next_scores, float *diff) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	float local_diff = 0;
	if(src < m) {
		local_diff += fabs(next_scores[src] - cur_scores[src]);
		cur_scores[src] = next_scores[src];
		next_scores[src] = (1.0f - kDamp) / m;
	}
	float block_sum = BlockReduce(temp_storage).Sum(local_diff);
	if(threadIdx.x == 0) atomicAdd(diff, block_sum);
}

void PRSolver(int m, int nnz, int *in_row_offsets, int *in_column_indices, int *h_row_offsets, int *h_column_indices, int *h_degree, ScoreT *h_scores) {
	int *d_row_offsets, *d_column_indices;
	ScoreT *d_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, m * sizeof(ScoreT)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, h_scores, m * sizeof(ScoreT), hipMemcpyHostToDevice));
	ScoreT *d_next_scores;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_next_scores, m * sizeof(ScoreT)));
	float *d_diff, h_diff;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_diff, sizeof(float)));

	Timer t;
	int iter = 0;
	const ScoreT base_score = (1.0f - kDamp) / m;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	initialize <<<nblocks, nthreads>>> (m, d_next_scores, base_score);
	CudaTest("initializing failed");
	printf("Launching CUDA PR solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	t.Start();
	do {
		++ iter;
		h_diff = 0;
		CUDA_SAFE_CALL(hipMemcpy(d_diff, &h_diff, sizeof(float), hipMemcpyHostToDevice));
		scatter <<<nblocks, nthreads>>> (m, d_row_offsets, d_column_indices, d_scores, d_next_scores);
		CudaTest("solving kernel scatter failed");
		reduce <<<nblocks, nthreads>>> (m, d_scores, d_next_scores, d_diff);
		CudaTest("solving kernel reduce failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_diff, d_diff, sizeof(float), hipMemcpyDeviceToHost));
		//printf("iteration=%d, diff=%f\n", iter, h_diff);
		printf(" %2d    %lf\n", iter, h_diff);
		//CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	} while (h_diff > EPSILON && iter < MAX_ITER);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();
	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", PR_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, m * sizeof(ScoreT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_scores));
	CUDA_SAFE_CALL(hipFree(d_next_scores));
	CUDA_SAFE_CALL(hipFree(d_diff));
	return;
}
