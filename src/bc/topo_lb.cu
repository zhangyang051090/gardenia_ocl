#include "hip/hip_runtime.h"
// Copyright (c) 2016, Xuhao Chen
#define BC_VARIANT "topo_lb"
#include "bc.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "worklistc.h"
#include "timer.h"
#include <vector>
#include <hipcub/hipcub.hpp>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

__global__ void initialize(int m, int source, ScoreT *scores, int *path_counts, int *depths, ScoreT *deltas, bool *visited, bool *expanded) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		scores[id] = 0;
		deltas[id] = 0;
		expanded[id] = false;
		if(id == source) {
			visited[id] = true;
			path_counts[id] = 1;
			depths[id] = 0;
		} else {
			visited[id] = false;
			path_counts[id] = 0;
			depths[id] = -1;
		}
	}
}

__device__ __forceinline__ void process_edge(int src, int depth, int edge, int *column_indices, int *path_counts, int *depths, bool *changed) {
	int dst = column_indices[edge];
	//assert(dst < m);
	if(depths[dst] == -1) {
		depths[dst] = depth;
		*changed = true;
	}
	if (depths[dst] == depth) {
		atomicAdd(&path_counts[dst], path_counts[src]);
	}
}

__device__ void expandByCta(int m, int *row_offsets, int *column_indices, int *path_counts, int *depths, int depth, bool *visited, bool *expanded, bool *changed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int vertex = id;
	__shared__ int owner;
	__shared__ int sh_vertex;
	owner = -1;
	int size = 0;
	if(vertex < m && visited[vertex] && !expanded[vertex]) {
		size = row_offsets[vertex + 1] - row_offsets[vertex];
	}
	while(true) {
		if(size > BLOCK_SIZE)
			owner = threadIdx.x;
		__syncthreads();
		if(owner == -1)
			break;
		__syncthreads();
		if(owner == threadIdx.x) {
			sh_vertex = vertex;
			expanded[id] = true;
			owner = -1;
			size = 0;
		}
		__syncthreads();
		int row_begin = row_offsets[sh_vertex];
		int row_end = row_offsets[sh_vertex + 1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
		for(int i = threadIdx.x; i < num; i += blockDim.x) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				process_edge(sh_vertex, depth, edge, column_indices, path_counts, depths, changed);
			}
		}
	}
}

__device__ __forceinline__ unsigned LaneId() {
	unsigned ret;
	asm("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

__device__ __forceinline__ void expandByWarp(int m, int *row_offsets, int *column_indices, int *path_counts, int *depths, int depth, bool *visited, bool *expanded, bool *changed) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned warp_id = threadIdx.x >> LOG_WARP_SIZE;
	unsigned lane_id = LaneId();
	__shared__ int owner[NUM_WARPS];
	__shared__ int sh_vertex[NUM_WARPS];
	owner[warp_id] = -1;
	int size = 0;
	int vertex = id;
	if(vertex < m && visited[vertex] && !expanded[vertex]) {
		size = row_offsets[vertex + 1] - row_offsets[vertex];
	}
	while(__any(size) >= WARP_SIZE) {
		if(size >= WARP_SIZE)
			owner[warp_id] = lane_id;
		if(owner[warp_id] == lane_id) {
			sh_vertex[warp_id] = vertex;
			expanded[id] = true;
			owner[warp_id] = -1;
			size = 0;
		}
		int winner = sh_vertex[warp_id];
		int row_begin = row_offsets[winner];
		int row_end = row_offsets[winner + 1];
		int neighbor_size = row_end - row_begin;
		int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
		for(int i = lane_id; i < num; i+= WARP_SIZE) {
			int edge = row_begin + i;
			if(i < neighbor_size) {
				process_edge(winner, depth, edge, column_indices, path_counts, depths, changed);
			}
		}
	}
}

// Shortest path calculation by forward BFS
__global__ void bc_forward(int m, int *row_offsets, int *column_indices, int *path_counts, int *depths, int depth, bool *changed, bool *visited, bool *expanded) {
	expandByCta(m, row_offsets, column_indices, path_counts, depths, depth, visited, expanded, changed);
	expandByWarp(m, row_offsets, column_indices, path_counts, depths, depth, visited, expanded, changed);
	typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	const int SCRATCHSIZE = BLOCK_SIZE;
	__shared__ BlockScan::TempStorage temp_storage;
	__shared__ int gather_offsets[SCRATCHSIZE];
	__shared__ int srcsrc[SCRATCHSIZE];
	gather_offsets[threadIdx.x] = 0;
	int neighbor_size = 0;
	int neighbor_offset = 0;
	int scratch_offset = 0;
	int total_edges = 0;
	if(src < m && visited[src] && !expanded[src]) { // visited but not expanded
		expanded[src] = true;
		neighbor_offset = row_offsets[src];
		neighbor_size = row_offsets[src+1] - neighbor_offset;
	}
	BlockScan(temp_storage).ExclusiveSum(neighbor_size, scratch_offset, total_edges);
	int done = 0;
	int neighbors_done = 0;
	while(total_edges > 0) {
		__syncthreads();
		int i;
		for(i = 0; neighbors_done + i < neighbor_size && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
			gather_offsets[scratch_offset + i - done] = neighbor_offset + neighbors_done + i;
			srcsrc[scratch_offset + i - done] = src;
		}
		neighbors_done += i;
		scratch_offset += i;
		__syncthreads();
		int edge = gather_offsets[threadIdx.x];
		if(threadIdx.x < total_edges) {
			process_edge(srcsrc[threadIdx.x], depth, edge, column_indices, path_counts, depths, changed);
		}
		total_edges -= BLOCK_SIZE;
		done += BLOCK_SIZE;
	}
}

// Dependency accumulation by back propagation
__global__ void bc_reverse(int num, int *row_offsets, int *column_indices, int start, int *frontiers, ScoreT *scores, int *path_counts, int *depths, int depth, ScoreT *deltas) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < num) {
		int src = frontiers[start + id];
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
		ScoreT delta_src = 0;
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if(depths[dst] == depth + 1) {
				delta_src += static_cast<ScoreT>(path_counts[src]) / 
					static_cast<ScoreT>(path_counts[dst]) * (1 + deltas[dst]);
			}
		}
		deltas[src] = delta_src;
		scores[src] += deltas[src];
	}
}

// Dependency accumulation by back propagation
__global__ void bc_reverse_lb(int num, int *row_offsets, int *column_indices, int start, int *frontiers, ScoreT *scores, int *path_counts, int *depths, int depth, ScoreT *deltas) {
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];
	__shared__ ScoreT sdata[BLOCK_SIZE + 16];                       // padded to avoid reduction conditionals

	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	for(int index = warp_id; index < num; index += num_warps) {
		int src = frontiers[start + index];
		// use two threads to fetch Ap[row] and Ap[row+1]
		// this is considerably faster than the straightforward version
		if(thread_lane < 2)
			ptrs[warp_lane][thread_lane] = row_offsets[src + thread_lane];
		const int row_begin = ptrs[warp_lane][0];                   //same as: row_start = row_offsets[row];
		const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = row_offsets[row+1];
		ScoreT sum = 0;
		for(int offset = row_begin + thread_lane; offset < row_end; offset += WARP_SIZE) {
			int dst = column_indices[offset];
			if(depths[dst] == depth + 1) {
				sum += static_cast<ScoreT>(path_counts[src]) / 
					static_cast<ScoreT>(path_counts[dst]) * (1 + deltas[dst]);
			}
		}
		// store local sum in shared memory
		sdata[threadIdx.x] = sum; __syncthreads();

		// reduce local sums to row sum
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();
		if (thread_lane == 0) {
			deltas[src] += sdata[threadIdx.x];
			scores[src] += deltas[src];
		}
	}
}

__global__ void bc_update(int m, int *depths, bool *visited, int *nitems, int *queue, int queue_len) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(depths[id] != -1 && !visited[id]) {
			visited[id] = true;
			int pos = atomicAdd(nitems, 1);
			queue[queue_len + pos] = id;
		}
	}
}

__global__ void bc_normalize(int m, ScoreT *scores, ScoreT max_score) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < m) {
		scores[tid] = scores[tid] / (max_score);
	}
}

void BCSolver(int m, int nnz, int source, int *h_row_offsets, int *h_column_indices, ScoreT *h_scores) {
	//print_device_info(0);
	int zero = 0;
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	ScoreT *d_scores, *d_deltas;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, sizeof(ScoreT) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_deltas, sizeof(ScoreT) * m));
	int *d_path_counts, *d_depths, *d_frontiers;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_path_counts, sizeof(int) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_depths, sizeof(int) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_frontiers, sizeof(int) * (m+1)));
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	int *d_nitems, h_nitems = 1;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_nitems, sizeof(int)));

	int depth = 0;
	vector<int> depth_index;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	initialize <<<nblocks, nthreads>>> (m, source, d_scores, d_path_counts, d_depths, d_deltas, d_visited, d_expanded);
	CudaTest("initializing failed");
	CUDA_SAFE_CALL(hipMemcpy(&d_frontiers[0], &source, sizeof(int), hipMemcpyHostToDevice));
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	const int nSM = deviceProp.multiProcessorCount;
	const int max_blocks_per_SM = maximum_residency(bc_reverse_lb, nthreads, 0);
	const int max_blocks = max_blocks_per_SM * nSM;
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	int frontiers_len = 0;
	depth_index.push_back(0);
	printf("Launching CUDA BC solver (%d CTAs/SM, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		depth++;
		h_changed = false;
		//printf("iteration=%d, frontire_size=%d\n", depth, h_nitems);
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_nitems, &zero, sizeof(int), hipMemcpyHostToDevice));
		frontiers_len += h_nitems;
		depth_index.push_back(frontiers_len);
		bc_forward<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_path_counts, d_depths, depth, d_changed, d_visited, d_expanded);
		CudaTest("solving bc_forward failed");
		bc_update <<<nblocks, nthreads>>> (m, d_depths, d_visited, d_nitems, d_frontiers, frontiers_len);
		CudaTest("solving bc_update failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(&h_nitems, d_nitems, sizeof(int), hipMemcpyDeviceToHost));
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	//printf("\nDone Forward BFS, starting back propagation (dependency accumulation)\n");
	for (int d = depth_index.size() - 2; d >= 0; d--) {
		h_nitems = depth_index[d+1] - depth_index[d];
		//thrust::sort(thrust::device, d_frontiers+depth_index[d], d_frontiers+depth_index[d+1]);
		//nblocks = (h_nitems - 1) / nthreads + 1;
		nblocks = std::min(max_blocks, DIVIDE_INTO(h_nitems, WARPS_PER_BLOCK));
		//printf("Reverse: depth=%d, frontier_size=%d\n", d, h_nitems);
		bc_reverse_lb<<<nblocks, nthreads>>>(h_nitems, d_row_offsets, d_column_indices, depth_index[d], d_frontiers, d_scores, d_path_counts, d_depths, d, d_deltas);
		CudaTest("solving kernel_reverse failed");
	}
	//CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, sizeof(ScoreT) * m, hipMemcpyDeviceToHost));
	//printf("\nStart calculating the maximum score\n");
	ScoreT *d_max_score;
	d_max_score = thrust::max_element(thrust::device, d_scores, d_scores + m);
	ScoreT h_max_score;
	CUDA_SAFE_CALL(hipMemcpy(&h_max_score, d_max_score, sizeof(ScoreT), hipMemcpyDeviceToHost));
	//h_max_score = *max_element(h_scores, h_scores+m);
	//for (int n = 0; n < m; n ++) h_scores[n] = h_scores[n] / h_max_score;
	//std::cout << "max_score = " << h_max_score << "\n";
	//printf("\nStart normalizing scores\n");
	nthreads = 512;
	nblocks = (m - 1) / nthreads + 1;
	bc_normalize<<<nblocks, nthreads>>>(m, d_scores, h_max_score);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [%s] = %f ms.\n", BC_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, sizeof(ScoreT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_path_counts));
	CUDA_SAFE_CALL(hipFree(d_depths));
	CUDA_SAFE_CALL(hipFree(d_deltas));
	CUDA_SAFE_CALL(hipFree(d_frontiers));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
}

