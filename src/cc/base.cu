#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define CC_VARIANT "topo_base"
#include "cc.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

/*
Gardenia Benchmark Suite
Kernel: Connected Components (CC)
Author: Xuhao Chen

Will return comp array labelling each vertex with a connected component ID
This CC implementation makes use of the Shiloach-Vishkin algorithm
*/
__global__ void scatter(int m, int *row_offsets, int *column_indices, CompT *comp, bool *changed) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m) {
		int comp_src = comp[src];
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1]; 
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			int comp_dst = comp[dst];
			if ((comp_src < comp_dst) && (comp_dst == comp[comp_dst])) {
				*changed = true;
				comp[comp_dst] = comp_src;
			}
		}
	}
}

__global__ void update(int m, int *row_offsets, int *column_indices, CompT *comp) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
	if(src < m) {
		while (comp[src] != comp[comp[src]]) {
			comp[src] = comp[comp[src]];
		}
	}
}

void CCSolver(int m, int nnz, int *h_row_offsets, int *h_column_indices, CompT *h_comp) {
	//print_device_info(0);
	int *d_row_offsets, *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	CompT *d_comp;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_comp, sizeof(CompT) * m));
	CUDA_SAFE_CALL(hipMemcpy(d_comp, h_comp, m * sizeof(CompT), hipMemcpyHostToDevice));
	bool h_changed, *d_changed;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));

	int iter = 0;
	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA BFS solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(h_changed), hipMemcpyHostToDevice));
		//printf("iteration=%d\n", iter);
		scatter<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_comp, d_changed);
		CudaTest("solving kernel scatter failed");
		update<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_comp);
		CudaTest("solving kernel update failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(h_changed), hipMemcpyDeviceToHost));
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", CC_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_comp, d_comp, sizeof(CompT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_changed));
}

