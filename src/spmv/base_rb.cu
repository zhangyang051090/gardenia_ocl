#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include <stdio.h>
#define SPMV_VARIANT "scalar"
#include "spmv.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

////////////////////////////////////////////////////////////////////////
// CSR SpMV kernels based on a scalar model (one thread per row)
///////////////////////////////////////////////////////////////////////
//
// spmv_csr_scalar_device
//   Straightforward translation of standard CSR SpMV to CUDA
//   where each thread computes y[i] += A[i,:] * x 
//   (the dot product of the i-th row of A with the x vector)
//
// spmv_csr_scalar_tex_device
//   Same as spmv_csr_scalar_device, except x is accessed via texture cache.
//

__global__ void spmv_csr_scalar_kernel(const int num_rows, const int * Ap,  const int * Aj,
		const ValueType * Ax, const ValueType * x, ValueType * y, float * value, 
		int * block, int * row_start) {

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < num_rows / 2) {
		float d0 = y[2*row];
		float d1 = y[2*row+1];
		for(int j = row_start[row]; j < row_start[row+1]; j ++){

			d0 += value[j * 4 + 0]*x[2*block[j]+0];
			d0 += value[j * 4 + 1]*x[2*block[j]+1];
			d1 += value[j * 4 + 2]*x[2*block[j]+0];
			d1 += value[j * 4 + 3]*x[2*block[j]+1];
	//		d0 += value[j][1]*x[block[j]+1];
	//		d1 += value[j][2]*x[block[j]+0];
	//		d1 += value[j][3]*x[block[j]+1];
		
		}
		y[2*row] = d0;	
		y[2*row+1] = d1;
	}


}
/*
__global__ void spmv_csr_scalar_kernel(const int num_rows, const int * Ap,  const int * Aj,
		const ValueType * Ax, const ValueType * x, ValueType * y) {
;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < num_rows) {
		for(int i = 0; i < num_rows; i ++)
			B[i*num_rows + row] = 0.0;
	}

	if(row < num_rows) {
		for(int j = 0; j < num_rows; j ++)	
			for(int offset = Ap[row]; offset < Ap[row+1]; offset ++)
				B[j*num_rows + Aj[offset]] = Ax[offset];
	}

	//for (int i = 0; i < num_rows*num_rows; i=i+4){
	if((row%4 == 0) && row < num_rows){
			sum0 += B[row] * x[row];			
			sum1 += B[row+num_rows] * x[row];
			sum0 += B[row] * x[row + 1];			
			sum1 += B[row+num_rows] * x[row+1];
	}
	
	sum0= atomic(sum0);
	sum1= atomic(sum1);
	if(row == 0)
		sum = sum0 + sum1;

	int block_total = BlockReduce(temp_storage).Sum(local_total);
	if(threadIdx.x == 0) atomicAdd(total, block_total);


	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int bm = 2*row;
	//if(((bm % 2) == 0) && (bm < num_rows)) {
	if(bm < num_rows) {
	  //   for(int i = 0; i < 2; i ++){
	//	ValueType sum = y[bm+i];
		ValueType sum = y[bm];
			colidx[nnz/2];
		b_Ap[bm] = Ap[bm] / 2;

		int row_begin_1 = Ap[bm];
		int row_end_1 = Ap[bm+1];
		for (int offset = row_begin_1; offset < row_end_1; offset ++){
			colidx[offset] = Aj[offset]/2;
			if(offset > 0 && colidx[offset]!=colidx[offset-1])
				block_size++;
		}

		int row_begin_2 = Ap[bm+1];
		int row_end_2 = Ap[bm+2];
		for (int offset = row_begin_2; offset < row_end_2; offset ++){
			colidx[offset] = Aj[offset]/2;
			if(offset > 0 && (colidx[offset]!=colidx[offset-1]) &&)
				block_size++;
		}	

		b_Ax[offset] = 0;
		b_Ax[offset] = Ax[offset];

		


		//row_begin = min(row_begin_1, row_begin_2)/2;
		//row_end = max(row_end_1, row_end_2)/2;

		for (int bm_offset = row_begin_1; bm_offset < row_end_2; bm_offset ++){
		for (int block = 0; block < num_rows; block ++){
			//block spmv
			//for(int offset = bm_offset; offset < bm_offset + 4; offset ++){
		   
			




			for(int i = 0; i < 4; i ++)
				bitmask[i]=0;

			if(Aj[bm_offset]%2==0 && row%2==0)bitmask[0]=1;
			else if(Aj[bm_offset]%2==1 && row%2==0)bitmask[1]=1;
			else if(Aj[bm_offset]%2==0 && row%2==1)bitmask[2]=1;
			else bitmask[3]=1;	
			
			sum0 += Ax[bm_offset]*bitmask[0] * x[2*colidx[bm_offset]];			
			sum1 += Ax[bm_offset+(bm_row_end - bm_row_begin)]*bitmask[1] * x[2*colidx[bm_offset]];
			sum0 += Ax[bm_offset]*bitmask[2] * x[2*colidx[bm_offset] + 1];			
			sum1 += Ax[bm_offset+(bm_row_end - bm_row_begin)]*bitmask[3] * x[2*colidx[bm_offset] + 1];			
		}
	//	y[bm+i] = sum;
		y[bm] = sum;
	  //    }
	}

	int bm = blockIdx.x * blockDim.x + threadIdx.x;
	//if(((bm % 2) == 0) && (bm < num_rows)) {
	if(bm < num_rows) {
	  //   for(int i = 0; i < 2; i ++){
	//	ValueType sum = y[bm+i];
		ValueType sum = y[bm];
		b_Ap[bm] = Ap[bm] / 2;
		int row_begin = Ap[bm];
		int row_end = Ap[bm+1];
		bm_row_begin = row_begin / 2;
		bm_row_end = row_end / 2;
		for (int bm_offset = bm_row_begin; bm_offset < bm_row_end; bm_offset ++){
			//block spmv
			//for(int offset = bm_offset; offset < bm_offset + 4; offset ++){			
			sum0 += Ax[bm_offset] * x[Aj[offset]];			
			sum1 += Ax[bm_offset+(bm_row_end - bm_row_begin)] * x[Aj[offset]];
			sum0 += Ax[bm_offset] * x[Aj[offset] + 1];			
			sum1 += Ax[bm_offset+(bm_row_end - bm_row_begin)] * x[Aj[offset] + 1];			
		}
	//	y[bm+i] = sum;
		y[bm] = sum;
	  //    }
	}



}
*/
void SpmvSolver(int num_rows, int nnz, int *h_Ap, int *h_Aj, ValueType *h_Ax, ValueType *h_x, ValueType *h_y, ValueType *h_value, int *h_block, int *h_row_start, int &num_block_all) { 
	//print_device_info(0);
	int *d_Ap, *d_Aj;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (num_rows + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(int), hipMemcpyHostToDevice));
	ValueType *d_Ax, *d_x, *d_y;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(ValueType) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(ValueType) * num_rows));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, sizeof(ValueType) * num_rows));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(ValueType), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, num_rows * sizeof(ValueType), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, num_rows * sizeof(ValueType), hipMemcpyHostToDevice));
	float *d_value;
	int *d_block, *d_row_start;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_value, num_block_all * 4 * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_block, num_block_all * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_start, (num_rows / 2 + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_value, h_value, num_block_all * 4 * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_block, h_block, num_block_all * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_row_start, h_row_start, (num_rows / 2 + 1) * sizeof(int), hipMemcpyHostToDevice));
	
	for(int j = 0; j < 300; j ++){
			printf("h_value: %f \n", h_value[j]);
	}


	int nthreads = BLOCK_SIZE;
	int nblocks = (num_rows - 1) / nthreads + 1;
	printf("Launching CUDA SpMV solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	spmv_csr_scalar_kernel <<<nblocks, nthreads>>> (num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y, d_value, d_block, 
		d_row_start);   
	CudaTest("solving failed");
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [%s] = %f ms.\n", SPMV_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, sizeof(ValueType) * num_rows, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
	CUDA_SAFE_CALL(hipFree(d_value));
	CUDA_SAFE_CALL(hipFree(d_block));
	CUDA_SAFE_CALL(hipFree(d_row_start));
}

