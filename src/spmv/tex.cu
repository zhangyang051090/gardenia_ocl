#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include <stdio.h>
#define SPMV_VARIANT "texture"
#include "spmv.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

// CSR SpMV kernels based on a scalar model (one thread per row)
//
// spmv_csr_scalar_tex_device
//   Same as spmv_csr_scalar_device, except x is accessed via texture cache.

/*
template <bool UseCache>
__inline__ __device__ float fetch_x(const int& i, const float * x) {
    if (UseCache)
        return tex1Dfetch(tex_x, i);
    else
        return x[i];
}
*/
texture<float,1> tex_x;
void bind_x(const float * x) { CUDA_SAFE_CALL(hipBindTexture(NULL, tex_x, x)); }
void unbind_x(const float * x) { CUDA_SAFE_CALL(hipUnbindTexture(tex_x)); }

__global__ void spmv_csr_scalar_kernel(const int num_rows, const int * Ap,  const int * Aj,
		const ValueType * Ax, const ValueType * x, ValueType * y) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < num_rows) {
		ValueType sum = y[row];
		int row_begin = Ap[row];
		int row_end = Ap[row+1];
		for (int offset = row_begin; offset < row_end; offset ++){
			sum += Ax[offset] * tex1Dfetch(tex_x, Aj[offset]);
		}
		y[row] = sum;
	}
}

void SpmvSolver(int num_rows, int nnz, int *h_Ap, int *h_Aj, ValueType *h_Ax, ValueType *h_x, ValueType *h_y) { 
	//print_device_info(0);
	Timer t;
	int *d_Ap, *d_Aj;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (num_rows + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(int), hipMemcpyHostToDevice));
	ValueType *d_Ax, *d_x, *d_y;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(ValueType) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(ValueType) * num_rows));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, sizeof(ValueType) * num_rows));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(ValueType), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, num_rows * sizeof(ValueType), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, num_rows * sizeof(ValueType), hipMemcpyHostToDevice));
	int nthreads = BLOCK_SIZE;
	int nblocks = (num_rows - 1) / nthreads + 1;
	printf("Launching CUDA SpMV solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	t.Start();
	bind_x(d_x);
	spmv_csr_scalar_kernel <<<nblocks, nthreads>>> (num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y);   
	CudaTest("solving failed");
	unbind_x(d_x);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [%s] = %f ms.\n", SPMV_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, sizeof(ValueType) * num_rows, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
}

