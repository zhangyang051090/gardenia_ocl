#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#include <stdio.h>
#include <algorithm>
#define SPMV_VARIANT "vector"
#include "spmv.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include "timer.h"

// CSR SpMV kernels based on a vector model (one vector per row)
//
// spmv_csr_vector_device
//   Each row of the CSR matrix is assigned to a vector.  The vector computes
//   y[i] = A[i,:] * x, i.e. the dot product of the i-th row of A with 
//   the x vector, in parallel.  This division of work implies that 
//   the CSR index and data arrays (Aj and Ax) are accessed in a contiguous
//   manner (but generally not aligned).  On GT200 these accesses are
//   coalesced, unlike kernels based on the one-row-per-thread division of 
//   work.  Since an entire vector is assigned to each row, many 
//   threads will remain idle when their row contains a small number 
//   of elements.  This code relies on implicit synchronization among 
//   threads in a vector.

texture<float,1> tex_x;
void bind_x(const float * x) { CUDA_SAFE_CALL(hipBindTexture(NULL, tex_x, x)); }
void unbind_x(const float * x) { CUDA_SAFE_CALL(hipUnbindTexture(tex_x)); }

template <int VECTORS_PER_BLOCK, int THREADS_PER_VECTOR>
__global__ void spmv_vector_kernel(int num_rows, const int * Ap,  const int * Aj, const ValueType * Ax, const ValueType * x, ValueType * y) {
	__shared__ ValueType sdata[VECTORS_PER_BLOCK * THREADS_PER_VECTOR + THREADS_PER_VECTOR / 2]; // padded to avoid reduction ifs
	__shared__ int ptrs[VECTORS_PER_BLOCK][2];

	const int thread_id	  = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (THREADS_PER_VECTOR-1);   // thread index within the vector
	const int vector_id   = thread_id   / THREADS_PER_VECTOR;     // global vector index
	const int vector_lane = threadIdx.x / THREADS_PER_VECTOR;     // vector index within the CTA
	const int num_vectors = VECTORS_PER_BLOCK * gridDim.x;        // total number of active vectors

	for(int row = vector_id; row < num_rows; row += num_vectors) {
		// use two threads to fetch Ap[row] and Ap[row+1]
		// this is considerably faster than the straightforward version
		if(thread_lane < 2)
			ptrs[vector_lane][thread_lane] = Ap[row + thread_lane];
		const int row_start = ptrs[vector_lane][0];                   //same as: row_start = Ap[row];
		const int row_end   = ptrs[vector_lane][1];                   //same as: row_end   = Ap[row+1];

		// compute local sum
		ValueType sum = 0;
		for(int offset = row_start + thread_lane; offset < row_end; offset += THREADS_PER_VECTOR)
			//sum += Ax[offset] * x[Aj[offset]];
			sum += Ax[offset] * tex1Dfetch(tex_x, Aj[offset]);

		// reduce local sums to row sum
		sdata[threadIdx.x] = sum; __syncthreads();
		if (THREADS_PER_VECTOR > 16) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads(); 
		if (THREADS_PER_VECTOR >  8) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		if (THREADS_PER_VECTOR >  4) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		if (THREADS_PER_VECTOR >  2) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		if (THREADS_PER_VECTOR >  1) sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();

		// first thread writes vector result
		if (thread_lane == 0)
			y[row] += sdata[threadIdx.x];
	}
}

size_t nSM;
template <int THREADS_PER_VECTOR>
void spmv_vector(int num_rows, int *d_Ap, int *d_Aj, ValueType *d_Ax, ValueType *d_x, ValueType *d_y) {
	const int VECTORS_PER_BLOCK = BLOCK_SIZE / THREADS_PER_VECTOR;
	//const size_t max_blocks_per_SM = maximum_residency(spmv_vector_kernel<VECTORS_PER_BLOCK, THREADS_PER_VECTOR>, BLOCK_SIZE, 0);
	//const size_t max_blocks = max_blocks_per_SM * nSM;
	const int nblocks = std::min(MAX_BLOCKS, DIVIDE_INTO(num_rows, VECTORS_PER_BLOCK));
	//printf("Launching CUDA SpMV solver (%ld CTAs, %d threads/CTA) ...\n", nblocks, BLOCK_SIZE);
	//printf("vector size: %d\n", THREADS_PER_VECTOR);
	spmv_vector_kernel<VECTORS_PER_BLOCK, THREADS_PER_VECTOR> <<<nblocks, BLOCK_SIZE>>>(num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y);
	CudaTest("solving failed");
}

void SpmvSolver(int num_rows, int nnz, int *h_Ap, int *h_Aj, ValueType *h_Ax, ValueType *h_x, ValueType *h_y) { 
	//print_device_info(0);
	int *d_Ap, *d_Aj;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ap, (num_rows + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Aj, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_Ap, h_Ap, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_Aj, h_Aj, nnz * sizeof(int), hipMemcpyHostToDevice));
	ValueType *d_Ax, *d_x, *d_y;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_Ax, sizeof(ValueType) * nnz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, sizeof(ValueType) * num_rows));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, sizeof(ValueType) * num_rows));
	CUDA_SAFE_CALL(hipMemcpy(d_Ax, h_Ax, nnz * sizeof(ValueType), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, num_rows * sizeof(ValueType), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, num_rows * sizeof(ValueType), hipMemcpyHostToDevice));

	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
	nSM = deviceProp.multiProcessorCount;
	int nnz_per_row = nnz / num_rows;
	printf("Launching CUDA SpMV solver (%d threads/CTA) ...\n", BLOCK_SIZE);

	Timer t;
	t.Start();
	bind_x(d_x);
	if (nnz_per_row <=  2) spmv_vector<2>(num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y);
	else if (nnz_per_row <=  4) spmv_vector<4>(num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y);
	else if (nnz_per_row <=  8) spmv_vector<8>(num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y);
	else if (nnz_per_row <= 16) spmv_vector<16>(num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y);
	else spmv_vector<32>(num_rows, d_Ap, d_Aj, d_Ax, d_x, d_y);
	unbind_x(d_x);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [%s] = %f ms.\n", SPMV_VARIANT, t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, sizeof(ValueType) * num_rows, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_Ap));
	CUDA_SAFE_CALL(hipFree(d_Aj));
	CUDA_SAFE_CALL(hipFree(d_Ax));
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
}

